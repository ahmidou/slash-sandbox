#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2010-2011, NVIDIA Corporation
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above copyright
 *     notice, this list of conditions and the following disclaimer in the
 *     documentation and/or other materials provided with the distribution.
 *   * Neither the name of NVIDIA Corporation nor the
 *     names of its contributors may be used to endorse or promote products
 *     derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <nih/bvh/cuda/lbvh_builder.h>
#include <nih/sampling/random.h>
#include <nih/time/timer.h>
#include <nih/basic/cuda_domains.h>
#include <nih/bvh/bvh_tree.h>
#include <nih/tree/cuda/reduce.h>

namespace nih {

struct bbox_functor
{
    NIH_HOST_DEVICE Bbox4f operator() (
        const Vector4f op1,
        const Vector4f op2) const
    {
        Bbox4f result;
        result.insert( op1 );
        result.insert( op2 );
        return result;
    }
    NIH_HOST_DEVICE Bbox4f operator() (
        const Bbox4f op1,
        const Bbox4f op2) const
    {
        Bbox4f result;
        result.insert( op1 );
        result.insert( op2 );
        return result;
    }
};

void lbvh_test()
{
    fprintf(stderr, "lbvh test... started\n");

    const uint32 n_points = 4*1024*1024;
    const uint32 n_tests = 100;

    thrust::host_vector<Vector4f> h_points( n_points );

    Random random;
    for (uint32 i = 0; i < n_points; ++i)
        h_points[i] = Vector4f( random.next(), random.next(), random.next(), 1.0f );

    thrust::device_vector<Vector4f> d_points( h_points );
    thrust::device_vector<Vector4f> d_unsorted_points( h_points );

    thrust::device_vector<Bvh_node> bvh_nodes;
    thrust::device_vector<uint2>    bvh_leaves;
    thrust::device_vector<uint32>   bvh_index;

    cuda::LBVH_builder<uint64> builder( bvh_nodes, bvh_leaves, bvh_index );

    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );

    float time = 0.0f;

    for (uint32 i = 0; i <= n_tests; ++i)
    {
        d_points = d_unsorted_points;
        hipDeviceSynchronize();

        float dtime;
        hipEventRecord( start, 0 );

        builder.build(
            Bbox3f( Vector3f(0.0f), Vector3f(1.0f) ),
            d_points.begin(),
            d_points.end(),
            16u );

        hipEventRecord( stop, 0 );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &dtime, start, stop );

        if (i) // skip the first run
            time += dtime;
    }
    time /= 1000.0f * float(n_tests);

    hipEventDestroy( start );
    hipEventDestroy( stop );

    fprintf(stderr, "lbvh test... done\n");
    fprintf(stderr, "  time       : %f ms\n", time * 1000.0f );
    fprintf(stderr, "  points/sec : %f M\n", (n_points / time) / 1.0e6f );

    fprintf(stderr, "  nodes  : %u\n", builder.m_node_count );
    fprintf(stderr, "  leaves : %u\n", builder.m_leaf_count );
    for (uint32 level = 0; level < 60; ++level)
        fprintf(stderr, "  level %u : %u nodes\n", level, builder.m_levels[level+1] - builder.m_levels[level] );

    fprintf(stderr, "lbvh bbox reduction test... started\n");

    Bvh_tree<breadth_first_tree,device_domain> bvh(
        thrust::raw_pointer_cast( &bvh_nodes.front() ),
        builder.m_leaf_count,
        thrust::raw_pointer_cast( &bvh_leaves.front() ),
        60u,
        builder.m_levels );

    thrust::device_vector<Bbox4f> d_leaf_bboxes( builder.m_leaf_count );
    thrust::device_vector<Bbox4f> d_node_bboxes( builder.m_node_count );

    hipEventCreate( &start );
    hipEventCreate( &stop );

    time = 0.0f;

    for (uint32 i = 0; i <= n_tests; ++i)
    {
        float dtime;
        hipEventRecord( start, 0 );

        cuda::tree_reduce(
            bvh,
            thrust::raw_pointer_cast( &d_points.front() ),
            thrust::raw_pointer_cast( &d_leaf_bboxes.front() ),
            thrust::raw_pointer_cast( &d_node_bboxes.front() ),
            bbox_functor(),
            Bbox4f() );

        hipEventRecord( stop, 0 );
        hipEventSynchronize( stop );
        hipEventElapsedTime( &dtime, start, stop );

        if (i) // skip the first run
            time += dtime;
    }
    time /= 1000.0f * float(n_tests);

    hipEventDestroy( start );
    hipEventDestroy( stop );

    fprintf(stderr, "lbvh bbox reduction test... done\n");
    fprintf(stderr, "  time       : %f ms\n", time * 1000.0f );
    fprintf(stderr, "  points/sec : %f M\n", (n_points / time) / 1.0e6f );
}

} // namespace nih

